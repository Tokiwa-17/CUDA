#include "../include/config.cuh"
#include "../include/til.cuh"
#include "../include/matrixTranspose.cuh"

void matrixTranspose(int *A, int *B, int m, int n){

    // 输入矩阵A, 输出矩阵B = A ^ T.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    printMatrix(A, m, n);
    printMatrix(B, m, n);

    int *d_A, *d_B;
    CHECK(hipMalloc((void **)&d_A, sizeof(int) * (m * n)));
    CHECK(hipMalloc((void **)&d_B, sizeof(int) * (m * n)));
    
    CHECK(hipMemcpy(d_A, A, sizeof(int) * (m * n), hipMemcpyHostToDevice));

    float *f_A, *f_B;
    CHECK(hipMalloc((void **)&f_A, sizeof(float) * (m * n)));
    CHECK(hipMalloc((void **)&f_B, sizeof(float) * (m * n)));

    dim3 block(BLOCK_SIZE, BLOCK_SIZE), grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);

    intPtrToFloatPtr<<<grid, block>>>(d_A, f_A, m, n);

    float alpha = 1.f, beta = 0.f;

    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, f_A, m, &beta, f_B, m, f_B, m);

    /*********DEBUG********
    float* testA, *testB;
    testA = (float*)malloc(sizeof(float) * (m * n));
    testB = (float*)malloc(sizeof(float) * (m * n));
    CHECK(hipMemcpy(testA, f_A, sizeof(float) * (m * n), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(testB, f_B, sizeof(float) * (m * n), hipMemcpyDeviceToHost));
    printMatrix(testA, m, n);
    printMatrix(testB, m, n);
    ***********************/

    floatPtrToIntPtr<<<grid, block>>>(f_B, d_B, m, n);

    CHECK(hipMemcpy(B, d_B, sizeof(int) * (m * n), hipMemcpyDeviceToHost));

    hipFree(d_A);
    hipFree(d_B);
    hipFree(f_A);
    hipFree(f_B);
}
