#include "hip/hip_runtime.h"
#include "../include/config.cuh"
#include "../include/til.cuh"
#include "../include/matrixTranspose.cuh"

__global__ void intPtrToFloatPtr(int *in, float* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = in[idx] * 1.0f;
}

__global__ void floatPtrToIntPtr(float *in, int* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = (int)in[idx];
}

void matrixTranspose(int *A, int *B, int m, int n){

    // 输入矩阵A, 输出矩阵B = A ^ T.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int *d_A, *d_B;
    CHECK(hipMalloc((void **)&d_A, sizeof(int) * (m * n)));
    CHECK(hipMalloc((void **)&d_B, sizeof(int) * (m * n)));
    
    CHECK(hipMemcpy(d_A, A, sizeof(int) * (m * n), hipMemcpyHostToDevice));

    float *f_A, *f_B;
    CHECK(hipMalloc((void **)&f_A, sizeof(float) * (m * n)));
    CHECK(hipMalloc((void **)&f_B, sizeof(float) * (m * n)));

    dim3 block(m, 1), grid(n, 1);

    intPtrToFloatPtr<<<grid, block>>>(d_A, f_A, m, n);

    float alpha = 0.f, beta = 1.f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, f_A, m, &beta, f_B, m);

    floatPtrToIntPtr<<<grid, block>>>(f_B, d_B, m, n);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(f_A);
    hipFree(f_B);
}
