#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/gpuMatrixCublas.cuh"
#include "../include/config.cuh"
#include "../include/til.cuh"

__global__ void intPtrToFloatPtr(int *in, float* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = in[idx] * 1.0f;
}

void gpuMatrixCublas(int* A, int* B, int* C, int lda, int ldb, int ldc,
                     int m, int n, int k, float alpha, float beta){
    
    //cudaStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float* f_A, *f_B, *f_C;
    hipMalloc((void**)&f_A, sizeof(int) * (m * n));
    hipMalloc((void**)&f_B, sizeof(int) * (n * k));
    hipMalloc((void**)&f_C, sizeof(int) * (m * k));

    dim3 block(m, 1), grid(n, 1);
    intPtrToFloatPtr<<<grid, block>>>(A, f_A, m, n);
    intPtrToFloatPtr<<<grid, block>>>(B, f_B, n, k);
    hipDeviceSynchronize();

    double iStart = cpuSecond();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, 
        &alpha, f_B, ldb, f_A, lda, &beta, f_C, ldc);
    double iElaps = cpuSecond() - iStart;
    printf("gpu Matrix Benchmark(Cublas)\telapsed %f sec.\n", iElaps);
}
