#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/gpuMatrixCublas.cuh"
#include "../include/config.cuh"
#include "../include/til.cuh"

void gpuMatrixCublas(float* A, float* B, float* C, int lda, int ldb, int ldc,
                     int m, int n, int k, float alpha, float beta){
    
    //cudaStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    /*if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("CUBLAS initialization error!\n");
        return;
    }*/
    
    double iStart = cpuSecond();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, 
        &alpha, B, ldb, A, lda, &beta, C, ldc);
    double iElaps = cpuSecond() - iStart;
    printf("gpu Matrix Benchmark\t\telapsed %f sec.\n", iElaps);
}
