#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "../include/gpuMatrixCublas.cuh"
#include "../include/config.cuh"
#include "../include/til.cuh"
using namespace std;

__global__ void intPtrToFloatPtr(int *in, float* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = in[idx] * 1.0f;
}

__global__ void floatPtrToIntPtr(float *in, int* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = (int)in[idx];
}

void gpuMatrixCublas(int* A, int* B, int* C, int lda, int ldb, int ldc,
                     int m, int n, int k, float alpha, float beta){
    
    //cudaStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int *d_A, *d_B;
    CHECK(hipMalloc((void**)&d_A, sizeof(int) * (m * n)));
    CHECK(hipMalloc((void**)&d_B, sizeof(int) * (n * k)));
    CHECK(hipMemcpy(d_A, A, sizeof(int) * (m * n), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, sizeof(int) * (n * k), hipMemcpyHostToDevice));

    float* f_A, *f_B, *f_C;
    int* f_odata;
    CHECK(hipMalloc((void**)&f_A, sizeof(float) * (m * n)));
    CHECK(hipMalloc((void**)&f_B, sizeof(float) * (n * k)));
    CHECK(hipMalloc((void**)&f_C, sizeof(float) * (m * k)));
    CHECK(hipMalloc((void**)&f_odata, sizeof(int) * (m * k)));

    int *f_odataCopy;
    f_odataCopy = (int*)malloc(sizeof(int) * (m * k));

    dim3 block(m, 1), grid(n, 1);

    intPtrToFloatPtr<<<grid, block>>>(d_A, f_A, m, n);
    intPtrToFloatPtr<<<grid, block>>>(d_B, f_B, n, k);
    hipDeviceSynchronize();

    /********DEBUG********
    float* test;
    test = (float*)malloc(sizeof(float) * (m * n));
    CHECK(hipMemcpy(test, f_A, sizeof(float) * (m * n), hipMemcpyDeviceToHost));
    printMatrix(A, m, n);
    printMatrix(test, m, n);
    *********************/

    double iStart = cpuSecond();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, 
        &alpha, f_B, ldb, f_A, lda, &beta, f_C, ldc);
    //cublas 列主序所以调换f_B和f_A的位置
    double iElaps = cpuSecond() - iStart;
    printf("gpu Matrix Benchmark(Cublas)\t\telapsed %f sec.\n", iElaps);

    floatPtrToIntPtr<<<grid, block>>>(f_C, f_odata, m, k);
    hipblasGetMatrix(m, k, sizeof(float), f_odata, m, f_odataCopy, m);
    checkResult(C, f_odataCopy, m);
    
    
    hipFree(f_A);
    hipFree(f_B);
    hipFree(f_C);
    hipFree(f_odata);
    free(f_odataCopy);
}
