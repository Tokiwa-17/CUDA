#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "../include/gpuMatrixCublas.cuh"
#include "../include/config.cuh"
#include "../include/til.cuh"
using namespace std;

__global__ void intPtrToFloatPtr(int *in, float* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = in[idx] * 1.0f;
}

__global__ void floatPtrToIntPtr(float *in, int* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = (int)in[idx];
}

void gpuMatrixCublas(int* A, int* B, int* C, int lda, int ldb, int ldc,
                     int m, int n, int k, float alpha, float beta){
    
    //cudaStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int *d_A, *d_B;
    CHECK(hipMalloc((void**)&d_A, sizeof(int) * (m * n)));
    CHECK(hipMalloc((void**)&d_B, sizeof(int) * (n * k)));
    CHECK(hipMemcpy(d_A, A, sizeof(int) * (m * n), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, sizeof(int) * (n * k), hipMemcpyHostToDevice));

    float* f_A, *f_B, *f_C, *f_odata;
    CHECK(hipMalloc((void**)&f_A, sizeof(float) * (m * n)));
    CHECK(hipMalloc((void**)&f_B, sizeof(float) * (n * k)));
    CHECK(hipMalloc((void**)&f_C, sizeof(float) * (m * k)));
    f_odata = (float*)malloc(sizeof(float) * (m * k));

    int *f_odataCopy;
    f_odataCopy = (int*)malloc(sizeof(int) * (m * k));

    dim3 block(m, 1), grid(n, 1);

    intPtrToFloatPtr<<<grid, block>>>(d_A, f_A, m, n);
    intPtrToFloatPtr<<<grid, block>>>(d_B, f_B, n, k);
    hipDeviceSynchronize();

    /********DEBUG********
    float* test;
    test = (float*)malloc(sizeof(float) * (m * n));
    CHECK(hipMemcpy(test, f_A, sizeof(float) * (m * n), hipMemcpyDeviceToHost));
    printMatrix(A, m, n);
    printMatrix(test, m, n);
    *********************/

    double iStart = cpuSecond();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, 
        &alpha, f_B, ldb, f_A, lda, &beta, f_C, ldc);
    //cublas 列主序所以调换f_B和f_A的位置
    double iElaps = cpuSecond() - iStart;
    printf("gpu Matrix Benchmark(Cublas)\t\telapsed %f sec.\n", iElaps);

    hipblasGetMatrix(m, k, sizeof(float), f_C, m, f_odata, m);
    //floatPtrToIntPtr<<<grid, block>>>(f_odata, f_odataCopy, m, k);
    printMatrix(f_odata, m, k);
    printMatrix(C, m, k);
    checkResult(C, f_odataCopy, m);
    
    
    hipFree(f_A);
    hipFree(f_B);
    hipFree(f_C);
    free(f_odata);
    free(f_odataCopy);
}
