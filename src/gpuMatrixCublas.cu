#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/gpuMatrixCublas.cuh"
#include "../include/config.cuh"
#include "../include/til.cuh"

void gpuMatrixCublas(float* d_A, float* d_B, float* d_C, int lda, int ldb, int ldc,
                     int m, int n, int k, double alpha, double beta){
    
    cudaStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    
    if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("CUBLAS initialization error!\n");
        return;
    }
    double iStart = cpuSecond();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, 
        &alpha, B, ldb, A, lda, &beta, C, ldc);
    double iElaps = cpuSecond() - iStart;
    printf("gpu Matrix Benchmark\t\telapsed %f sec.\n", iElaps);
}
