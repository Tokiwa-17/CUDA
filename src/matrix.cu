#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matrix.cuh"

/*
*********************************************************************
function name : gpuMatrixMul
description : multiplication of two matrix
parameters :
    &d_A GPU device pointer to a (m, n) matrix(A)
    &d_B GPU device pointer to a (n, k) matrix(B)
    &d_C GPU device output pointer to a (m, k) matrix(C)
return: none
*********************************************************************
*/
__global__ void gpuMatrixMul(int* d_A, int* d_B, int* d_C, int m, int n, int k) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    int sum = 0;
    if (row < m && col < k) {
        for (int i = 0;i < n;i++)
            sum += d_A[row * n + i] * d_B[i * k + col];
        d_C[row * k + col] = sum;
    }
}

/*
*********************************************************************
function name: cpuMatrixMul
description: Multiplication two matrix in CPU.
parameters: 
    &h_A CPU host pointer to a (m, n) matrix (A)
    &h_B CPU host pointer to a (n, k) matrix (B)
    &h_C CPU host output pointer to a (m, k) matrix (C) 
    to store the result
return: none
*********************************************************************
*/
void cpuMatrixMul(int *h_A, int * h_B, int* h_C, int m, int n, int k){
    for(int i = 0;i < m;i++)
        for(int j = 0;j < k;j++){
            int sum = 0;
            for(int l = 0;l < n;l++)
                sum += h_A[i * n + l] * h_B[l * k + j];
            h_C[i * k + j] = sum;
        }
}
