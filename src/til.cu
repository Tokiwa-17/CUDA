#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/til.cuh"
#include "../include/config.cuh"

void cpuMatrixMul(int *h_A, int * h_B, int* h_C, int m, int n, int k){
    for(int i = 0;i < m;i++)
        for(int j = 0;j < k;j++){
            int sum = 0;
            for(int l = 0;l < n;l++)
                sum += h_A[i * n + l] * h_B[l * k + j];
            h_C[i * k + j] = sum;
        }
}

#ifdef _WIN32
int gettimeofday(struct timeval* tp, void* tzp){
    time_t clock;
    struct tm tm;
    SYSTEMTIME wtm;
    GetLocalTime(&wtm);
    tm.tm_year = wtm.wYear - 1900;
    tm.tm_mon = wtm.wMonth - 1;
    tm.tm_mday = wtm.wDay;
    tm.tm_hour = wtm.wHour;
    tm.tm_min = wtm.wMinute;
    tm.tm_sec = wtm.wSecond;
    tm.tm_isdst = -1;
    clock = mktime(&tm);
    tp->tv_sec = clock;
    tp->tv_usec = wtm.wMilliseconds * 1000;
    return (0);
}
#endif

double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

void initialData(float* ip, int size){
    //generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0;i < size;i++){
        ip[i] = (float)(rand() & 0xffff) / 1000.0f;
    }
}

void initialDataInt(int* ip, int size){
    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++){
        ip[i] = int((rand() % 100) - 50);
    }
}

void printMatrix(int* C, const int nx, const int ny) {
    int* ic = C;
    printf("Matrix<%d,%d>:\n", ny, nx);
    for (int i = 0;i < ny;i++) {

        for (int j = 0;j < nx;j++) {
            printf("%6d ", ic[j]);
        }
        ic += nx;
        printf("\n");
    }
}

void printMatrix(float* C, const int nx, const int ny){
    float* ic = C;
    printf("Matrix<%d,%d>:\n", ny, nx);
    for (int i = 0;i < ny;i++) {

        for (int j = 0;j < nx;j++) {
            printf("%6f ", ic[j]);
        }
        ic += nx;
        printf("\n");
    }
}

void initDevice(int devNum){
    int dev = devNum;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

}

void checkResult(int* hostRef, int* gpuRef, const int N){
    double epsilon = 1.0E-8;
    for (int i = 0;i < N;i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            printf("Results do not match!\n");
            printf("%d(hostRef[%d] )!= %d(gpuRef[%d])\n", hostRef[i], i, gpuRef[i], i);
            return;
        }
    }
    printf("Check result success!\n");
}

/*__global__ void intPtrToFloatPtr(int *in, float* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = in[idx] * 1.0f;
}


__global__ void floatPtrToIntPtr(float *in, int* out, unsigned int m, unsigned int n){
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;

    out[idx] = (int)in[idx];
}*/

__global__ void intPtrToFloatPtr(int *in, float *out, unsigned int m, unsigned int n){
    unsigned g_X = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned g_Y = blockIdx.y * blockDim.y + threadIdx.y;
    if(g_X < m && g_Y < n)
        out[g_Y * n + g_X] = in[g_Y * n + g_X] * 1.f;
}

__global__ void floatPtrToIntPtr(float *in, int *out, unsigned int m, unsigned int n){
    unsigned g_X = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned g_Y = blockIdx.y * blockDim.y + threadIdx.y;
    if(g_X < m && g_Y < n)
        out[g_Y * n + g_X] = (int)in[g_Y * n + g_X];
}

