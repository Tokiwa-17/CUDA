#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/matrixNaive.cuh"

/*
*********************************************************************
function name : gpuMatrixMul
description : multiplication of two matrix
parameters :
    &d_A GPU device pointer to a (m, n) matrix(A)
    &d_B GPU device pointer to a (n, k) matrix(B)
    &d_C GPU device output pointer to a (m, k) matrix(C)
return: none
*********************************************************************
*/
template<typename T>
__global__ void matrixNaive::gpuMatrixMul(T* d_A, T* d_B, T* d_C, int m, int n, int k) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    T sum = 0;
    if (row < m && col < k) {
        for (int i = 0;i < n;i++)
            sum += d_A[row * n + i] * d_B[i * k + col];
        d_C[row * k + col] = sum;
    }
}


