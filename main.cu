// Include C++ header files.
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "./include/config.cuh"
#include "./include/til.cuh"
#include "./include/matrixNaive.cuh"
#include "./include/matrixTile.cuh"
#include "./include/matrixTileWPT.cuh"
#include "./include/matrixTranspose.cuh"
#include "./include/matrixComOpt.cuh"
#include "./include/cpuMatrixStrassen.cuh"
//#include "./include/cublas.cuh"

// Include local CUDA header files.

int main(int argc, char ** argv){
    // set up device
    int dev = 0;
    initDevice(dev);

    // input m, n, k
    int m = 320, n = 320, k = 320;
    if(argc > 1) m = atoi(argv[1]);
    if(argc > 2) n = atoi(argv[2]);
    if(argc > 3) k = atoi(argv[3]);

    // Allocate memory space on the host
    int *h_A = (int*)malloc(sizeof(int) * (m * n));
    int *h_B = (int*)malloc(sizeof(int) * (n * k));
    int *h_C = (int*)malloc(sizeof(int) * (m * k));
    int *h_odata = (int*)malloc(sizeof(int) * (m * k));

    // Initialize 
    initialDataInt(h_A, m * n);
    initialDataInt(h_B, n * k);

    // Allocate memory space on the device
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(int) * (m * n));
    hipMalloc((void**)&d_B, sizeof(int) * (n * k));
    hipMalloc((void**)&d_C, sizeof(int) * (m * k));

    // Copy matrix A and B from host to device memory
    hipMemcpy(d_A, h_A, sizeof(int) * (m * n), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(int) * (n * k), hipMemcpyHostToDevice);

    // CPU Matrix multiplication
    double iStart = cpuSecond();
    cpuMatrixMul(h_A, h_B, h_C, m, n, k);
    double iElaps = cpuSecond() - iStart;   
    printf("cpu Matrix multiplication\t\telapsed %f sec.\n", iElaps);

    // CPU Matrix multiplication by Strassen
    /*cpuMatrixStrassen Strassen(h_A, h_B, m);
    double iStart = cpuSecond();
    ;
    cpuMatrixMul(h_A, h_B, h_C, m, n, k);
    double iElaps = cpuSecond() - iStart;   
    printf("cpu Matrix multiplication\t\telapsed %f sec.\n", iElaps);*/

    // GPU Matrix multiplication
    unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int gridCols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 grid(gridRows, gridCols);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    iStart = cpuSecond();
    gpuMatrixMul<< <grid, block >> > (d_A, d_B, d_C, m, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = cpuSecond() - iStart;
    CHECK(hipMemcpy(h_odata, d_C, sizeof(int) * (m * k), hipMemcpyDeviceToHost));

    printf("gpu Matrix multiplication\t\telapsed %f sec. <<<grid %d block "
        "%d>>>\n", iElaps, grid.x, block.x);

    // Check result
    checkResult(h_C, h_odata, m * k);

    //cublas(d_A, d_B, d_C, m, n, k);

    // GPU Matrix multiplication by tile
    block.x = TILE_SIZE, block.y = TILE_SIZE;
    grid.x = k / TILE_SIZE, grid.y = m / TILE_SIZE;
    if(grid.x == 0 || grid.y == 0){
        unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int gridCols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 grid(gridRows, gridCols);
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);

        iStart = cpuSecond();
        gpuMatrixMul<< <grid, block >> > (d_A, d_B, d_C, m, n, k);
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        iElaps = cpuSecond() - iStart;
    }
    else{
        iStart = cpuSecond();
        gpuMatrixMulTile<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        iElaps = cpuSecond() - iStart;
        CHECK(hipMemcpy(h_odata, d_C, sizeof(int) *(m * k), hipMemcpyDeviceToHost));
    }
    //printMatrix(h_odata, m, k);

    printf("gpu Matrix multiplication2\t\telapsed %f sec. <<<grid %d block "
    "%d>>>\n", iElaps, grid.x, block.x);
    checkResult(h_C, h_odata, m * k);

    // GPU Matrix multiplication by tile, optimized by WPT
    block.x = TILE_SIZE / WPT, block.y = TILE_SIZE;
    grid.x = k / TILE_SIZE, grid.y = m / TILE_SIZE;
    iStart = cpuSecond();
    gpuMatrixMulTileWPT<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = cpuSecond() - iStart;
    CHECK(hipMemcpy(h_odata, d_C, sizeof(int) *(m * k), hipMemcpyDeviceToHost));
    printf("gpu Matrix multiplication3\t\telapsed %f sec. <<<grid %d block "
    "%d>>>\n", iElaps, grid.x, block.x);
    checkResult(h_C, h_odata, m * k);

    // GPU Matrix multiplication by tile, optimized by WPT = 4
    block.x = TILE_SIZE / 4, block.y = TILE_SIZE;
    grid.x = k / TILE_SIZE, grid.y = m / TILE_SIZE;
    iStart = cpuSecond();
    gpuMatrixMulTileWPTop4<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = cpuSecond() - iStart;
    CHECK(hipMemcpy(h_odata, d_C, sizeof(int) *(m * k), hipMemcpyDeviceToHost));
    printf("gpu Matrix multiplication3(WPT = 4)\telapsed %f sec. <<<grid %d block "
    "%d>>>\n", iElaps, grid.x, block.x);
    checkResult(h_C, h_odata, m * k);

    // GPU Matrix multiplication by tile, optimized by WPT = 8
    block.x = TILE_SIZE / 8, block.y = TILE_SIZE;
    grid.x = k / TILE_SIZE, grid.y = m / TILE_SIZE;
    iStart = cpuSecond();
    gpuMatrixMulTileWPTop8<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = cpuSecond() - iStart;
    CHECK(hipMemcpy(h_odata, d_C, sizeof(int) *(m * k), hipMemcpyDeviceToHost));
    printf("gpu Matrix multiplication3(WPT = 8)\telapsed %f sec. <<<grid %d block "
    "%d>>>\n", iElaps, grid.x, block.x);
    checkResult(h_C, h_odata, m * k);

    // GPU Matrix multiplication by tile, optimized by Computational optimization4
    if(m > 32){
        block.x = TILE_SIZE, block.y = VEC_SIZE;
        grid.x = k / (TILE_SIZE * VEC_SIZE), grid.y = m / TILE_SIZE;
        //grid.x = (k + TILE_SIZE - 1) / TILE_SIZE, grid.y = (m + TILE_SIZE * VEC_SIZE - 1) / (TILE_SIZE * VEC_SIZE);
        iStart = cpuSecond();
        gpuMatrixComOpt<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        iElaps = cpuSecond() - iStart;
        CHECK(hipMemcpy(h_odata, d_C, sizeof(int) *(m * k), hipMemcpyDeviceToHost));
        printf("gpu Matrix multiplication4\t\telapsed %f sec. <<<grid %d block "
        "%d>>>\n", iElaps, grid.x, block.x);
        checkResult(h_C, h_odata, m * k);
    }

    // GPU Matrix multiplication by tile, optimized by Computational optimization8
    if(m > 64){
        block.x = TILE_SIZE, block.y = 8;
        grid.x = k / (TILE_SIZE * 8), grid.y = m / TILE_SIZE;
        iStart = cpuSecond();
        gpuMatrixComOpt8<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        iElaps = cpuSecond() - iStart;
        CHECK(hipMemcpy(h_odata, d_C, sizeof(int) *(m * k), hipMemcpyDeviceToHost));
        printf("gpu Matrix multiplication4\t\telapsed %f sec. <<<grid %d block "
        "%d>>>\n", iElaps, grid.x, block.x);
        checkResult(h_C, h_odata, m * k);
    }
    return 0;
}
