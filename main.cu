// Include C++ header files.
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "./include/til.cuh"
#include "./include/matrixNaive.cuh"
#include "./include/matrixTile.cuh"

// Include local CUDA header files.

int main(int argc, char ** argv){
    // set up device
    int dev = 0;
    initDevice(dev);

    // input m, n, k
    int m = 32, n = 32, k = 32;
    if(argc > 1) m = atoi(argv[1]);
    if(argc > 2) n = atoi(argv[2]);
    if(argc > 3) k = atoi(argv[3]);

    // Allocate memory space on the host
    int *h_A = (int*)malloc(sizeof(int) * (m * n));
    int *h_B = (int*)malloc(sizeof(int) * (n * k));
    int *h_C = (int*)malloc(sizeof(int) * (m * k));
    int *h_odata = (int*)malloc(sizeof(int) * (m * k));

    // Initialize 
    initialDataInt(h_A, m * n);
    initialDataInt(h_B, n * k);
    printMatrix(h_A, m, n);
    printMatrix(h_B, n, k);

    // Allocate memory space on the device
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(int) * (m * n));
    hipMalloc((void**)&d_B, sizeof(int) * (n * k));
    hipMalloc((void**)&d_C, sizeof(int) * (m * k));

    // Copy matrix A and B from host to device memory
    hipMemcpy(d_A, h_A, sizeof(int) * (m * n), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(int) * (n * k), hipMemcpyHostToDevice);

    // CPU Matrix multiplication
    double iStart = cpuSecond();
    cpuMatrixMul(h_A, h_B, h_C, m, n, k);
    double iElaps = cpuSecond() - iStart;   
    printf("cpu Matrix multiplication\t\telapsed %f sec.\n", iElaps);
    printMatrix(h_C, m, k);

    // GPU Matrix multiplication
    unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int gridCols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 grid(gridRows, gridCols);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    iStart = cpuSecond();
    gpuMatrixMul<< <grid, block >> > (d_A, d_B, d_C, m, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = cpuSecond() - iStart;
    CHECK(hipMemcpy(h_odata, d_C, sizeof(int) * (m * k), hipMemcpyDeviceToHost));

    printf("gpu Matrix multiplication\t\telapsed %f sec. <<<grid %d block "
        "%d>>>\n", iElaps, grid.x, block.x);

    // Check result
    checkResult(h_C, h_odata, m * k);

    // GPU Matrix multiplication by tile
    block.x = 16, block.y = 16;
    grid.x = k / 16, grid.y = m / 16;
    iStart = cpuSecond();
    gpuMatrixMulTile<<<grid, block>>>(d_A, d_B, d_C, m, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = cpuSecond() - iStart;
    CHECK(hipMemcpy(h_odata, d_C, sizeof(int) *(m * k), hipMemcpyDeviceToHost));

    printf("gpu Matrix multiplication2\t\telapsed %f sec. <<<grid %d block "
    "%d>>>\n", iElaps, grid.x, block.x);
    checkResult(h_C, h_odata, m * k);
    printMatrix(h_C, m, k);
    return 0;
}
