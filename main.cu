// Include C++ header files.
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "./include/freshman.cuh"
#include "./include/matrixNaive.cuh"

#define BLOCK_SIZE 32
// Include local CUDA header files.

/*
*********************************************************************
function name: cpuMatrixMul
description: Multiplication two matrix in CPU.
parameters: 
    &h_A CPU host pointer to a (m, n) matrix (A)
    &h_B CPU host pointer to a (n, k) matrix (B)
    &h_C CPU host output pointer to a (m, k) matrix (C) 
    to store the result
return: none
*********************************************************************
*/
void cpuMatrixMul(int *h_A, int * h_B, int* h_C, int m, int n, int k){
    for(int i = 0;i < m;i++)
        for(int j = 0;j < k;j++){
            int sum = 0;
            for(int l = 0;l < n;l++)
                sum += h_A[i * n + l] * h_B[l * k + j];
            h_C[i * k + j] = sum;
        }
}



int main(int argc, char ** argv){
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s \n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // input m, n, k
    int m = 100, n = 100, k = 100;
    if(argc > 1) m = atoi(argv[1]);
    if(argc > 2) n = atoi(argv[2]);
    if(argc > 3) k = atoi(argv[3]);

    // Allocate memory space on the host
    int *h_A = (int*)malloc(sizeof(int) * (m * n));
    int *h_B = (int*)malloc(sizeof(int) * (n * k));
    int *h_C = (int*)malloc(sizeof(int) * (m * k));
    int *h_odata = (int*)malloc(sizeof(int) * (m * k));

    // Initialize 
    initialDataInt(h_A, m * n);
    initialDataInt(h_B, n * k);
    initialDataInt(h_C, m * k);

    // Allocate memory space on the device
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(int) * (m * n));
    hipMalloc((void**)&d_B, sizeof(int) * (n * k));
    hipMalloc((void**)&d_C, sizeof(int) * (m * k));

    // Copy matrix A and B from host to device memory
    hipMemcpy(d_A, h_A, sizeof(int) * (m * n), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(int) * (n * k), hipMemcpyHostToDevice);

    unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int gridCols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 grid(gridRows, gridCols);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    // CPU Matrix multiplication
    double iStart = cpuSecond();
    cpuMatrixMul(h_A, h_B, h_C, m, n, k);
    double iElaps = cpuSecond() - iStart;   
    printf("cpu Matrix multiplication\t\telapsed %f sec.\n", iElaps);

    // GPU Matrix multiplication
    iStart = cpuSecond();
    gpuMatrixMul<int> << <grid, block >> > (d_A, d_B, d_C, m, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = cpuSecond() - iStart;
    CHECK(hipMemcpy(h_odata, d_C, sizeof(int) * (m * k), hipMemcpyDeviceToHost));

    printf("gpu Matrix multiplication\t\telapsed %f sec. <<<grid %d block "
        "%d>>>\n", iElaps, grid.x, block.x);

    // Check result
    checkResult(h_C, h_odata, m * k);
    return 0;
}
